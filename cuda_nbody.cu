#include "hip/hip_runtime.h"
#include "particule.h"

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include <float.h>
#include <sys/types.h>
#include <unistd.h>
#include <omp.h>

#define NBITER 1
#define BLOCKSIZE 16
#define GRIDDIM 32

static int SEEDED =0;
#define MIN(X, Y) (((X) < (Y)) ? (X) : (Y))

unsigned long mix(unsigned long a, unsigned long b, unsigned long c)
{
    a=a-b;  a=a-c;  a=a^(c >> 13);
    b=b-c;  b=b-a;  b=b^(a << 8);
    c=c-a;  c=c-b;  c=c^(b >> 13);
    a=a-b;  a=a-c;  a=a^(c >> 12);
    b=b-c;  b=b-a;  b=b^(a << 16);
    c=c-a;  c=c-b;  c=c^(b >> 5);
    a=a-b;  a=a-c;  a=a^(c >> 3);
    b=b-c;  b=b-a;  b=b^(a << 10);
    c=c-a;  c=c-b;  c=c^(b >> 15);
    return c;
}

pset * pset_alloc(int nb_par){
	pset * set = (pset *)malloc(sizeof(pset));
	if(set == NULL)
	{
		fprintf(stderr, "Can't allocate memory for the set creation.\n");
		exit(EXIT_FAILURE);
	}
	set->nb = nb_par;
	set->m = (double*)malloc(nb_par * sizeof(double));
	set->pos = (double*)malloc(3* nb_par * sizeof(double));
	set->spd = (double*)malloc(3* nb_par * sizeof(double));
	set->acc = (double*)malloc(3* nb_par * sizeof(double));
	set->force = (double*)malloc(3* nb_par * sizeof(double));
	return set;
}

void pset_free(pset * set){
	free(set->pos);
	free(set->spd);
	free(set->acc);
	free(set->m);
	free(set->force);
	free(set);
}

void pset_copy(pset * origin, pset * dest){
	int nb = origin-> nb;
	int sd = sizeof(double);
	dest->nb = origin->nb;
	memcpy(dest->m, origin->m  , nb*sd);
	memcpy(dest->acc, origin->acc, 3* nb*sd);
	memcpy(dest->spd, origin->spd, 3* nb*sd);
	memcpy(dest->pos, origin->pos, 3* nb*sd);
}

void pset_print(pset * set)
{
	int i;
	int size = set->nb;
	for (i = 0; i < size; ++i)
	{
		printf("#Particule numéro : %d, de masse %g\n", i, set->m[i]);
		printf("\tx:%g y:%g z:%g\n", set->pos[i], set->pos[i+ size], set->pos[i+ 2*size] );
		printf("\tvx:%g vy:%g vz:%g\n",set->spd[i], set->spd[i+ size], set->spd[i+ 2*size]);
		printf("\tax:%g ay:%g az:%g\n",set->acc[i], set->acc[i+ size], set->acc[i+ 2*size]);
	}
}

void seed()
{
	if(!SEEDED)
	{
		unsigned long seed = mix(clock(), time(NULL), getpid());
		srand(seed);
		SEEDED++;
	}
}

void pset_init_rand(pset * s)
{
	seed();
	int i;
	int size = s->nb;
	for (i = 0; i < size; i++)
	{
		s->m[i] = 1.0e10;
		s->pos[i] = MIN_RAND + rand()%(MAX_RAND-MIN_RAND);
		s->pos[i+size] = MIN_RAND + rand()%(MAX_RAND-MIN_RAND);
		s->pos[i+2*size] = MIN_RAND + rand()%(MAX_RAND-MIN_RAND);
		s->spd[i] = 0;
		s->spd[i+size] = 0;
		s->spd[i+2*size] = 0;
		s->acc[i] = 0;
		s->acc[i+size] = 0;
		s->acc[i+2*size] = 0;
	}
}
/* Calcule la vitesse de satellisation */
double v_orbit(double mass, double distance)
{
	return sqrt(CONST_GRAV*mass/distance);
}

void pset_init_orbit(pset *s)
{
	seed();
	double dmin= 200, distance;
	int size = s->nb;
	s->pos[0 ] = 0;
	s->pos[0 +size] = 0;
	s->pos[0 +2*size] = 0;
	s->spd[0 ] = 0;
	s->spd[0 +size] = 0;
	s->spd[0 +2*size] = 0;
	s->acc[0 ] = 0;
	s->acc[0 + size] = 0;
	s->acc[0 +2*size] = 0;

	s->m[0] = 1e10;

	for (int i = 1; i < size; ++i)
	{
		distance = dmin*i +  rand()% 50;
		s->m[i] = s->m[0] /20000;
		s->pos[i] = s->pos[0] -distance;
		s->pos[i+size] = 0;
		s->pos[i+2*size] = 0;
		s->spd[i] = 0;
		s->spd[i+size]= v_orbit(s->m[0], distance);
		s->spd[i+2*size]= 0;
		s->acc[i] = 0;
		s->acc[i+size] = 0;
		s->acc[i+2*size] = 0;
	}
}

/** Prend en argument les masses de deux particules, la distance entre
 * ces particules et retourne l'intensité de la force gravitationnelle
 * entre ces deux particules
 */


__device__ void distance(double x1, double y1, double z1 , double x2, double y2, double z2, double *res)
{
	*res = sqrt((x2-x1)*(x2-x1) + (y2-y1)*(y2-y1) + (z2-z1)*(z2-z1) );
}

__device__ void intensity(double m, double d, double * res)
{
	*res = (CONST_GRAV * m / (d*d*d));
}

__global__ void nbody(int* n, double* acc, double* spd, double* pos, double* m)
{
	unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
	
	int j;
	double d, inten1, inten2;
	int size = *n;
	double dt = 100.0;
	if(idx >  size)
		return;
	for (j = idx+1; j < size; ++j)
	{
		distance(pos[idx], pos[idx+ size], pos[idx+ 2*size], pos[j],
				 pos[j+ size], pos[j+ 2*size], &d);

		intensity(m[j], d, &inten1);
		acc[idx]+= inten1 *(pos[j] - pos[idx]); 
		acc[idx+size]+= inten1 *(pos[j+size] - pos[idx+size]);
		acc[idx+2*size]+= inten1 *(pos[j+2*size] - pos[idx+2*size]);

		intensity(m[idx], d, &inten2);
		acc[j]-= inten2 *(pos[j] - pos[idx]);  
		acc[j+size]-= inten2 *(pos[j+size] - pos[idx+size]);
		acc[j+2*size]-= inten2 *(pos[j+2*size] - pos[idx+2*size]);
	}

	pos[idx]+= dt* spd[idx] + dt*dt/2 * acc[idx];
	pos[idx + size]+= dt* spd[idx+ size] + dt*dt/2 * acc[idx+size];
	pos[idx + 2*size]+= dt* spd[idx+ 2*size] + dt*dt/2 * acc[idx+2*size];

	spd[idx]+= dt* acc[idx];
	spd[idx + size]+= dt* acc[idx+ size];
	spd[idx + 2*size]+= dt* acc[idx+ 2*size];
}


int main(int argc, char ** argv)
{
	if(argc != 2){
		fprintf(stderr, "Enter the number of particles\n");
		exit(EXIT_FAILURE);
	}
	int NBPAR = atoi(argv[1]);
	pset *s = pset_alloc(NBPAR);
	pset_init_orbit(s);

	/*pset_print(s);*/

	int* nb;
	double* acc, *spd, *pos, *m;
	hipMalloc((void**)&nb, 1*sizeof(int));
	hipMalloc((void**)&acc, 3*NBPAR*sizeof(double));
	hipMalloc((void**)&spd, 3*NBPAR*sizeof(double));
	hipMalloc((void**)&pos, 3*NBPAR*sizeof(double));
	hipMalloc((void**)&m, NBPAR*sizeof(double));

	hipMemcpy(nb, &s->nb, 1*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(acc, s->acc, 3*NBPAR*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(spd, s->spd, 3*NBPAR*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(pos, s->pos, 3*NBPAR*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(m, s->m, NBPAR*sizeof(double), hipMemcpyHostToDevice);

	dim3 dimBlock( BLOCKSIZE, 1 );
	dim3 dimGrid( GRIDDIM, GRIDDIM );

	FILE * fichier =fopen("datafile", "w+");
	/*fprintf(fichier, "#particule X Y Z\n");*/
	nbody<<< dimGrid, dimBlock >>>(nb, acc, spd, pos, m);
	hipMemcpy(s->pos, pos, 3*NBPAR*sizeof(double), hipMemcpyDeviceToHost);
	for (int i = 0; i < NBITER ; ++i)
	{
		nbody<<< dimGrid, dimBlock >>>(nb, acc, spd, pos, m);
		hipMemcpy(s->pos, pos, 3*NBPAR*sizeof(double), hipMemcpyDeviceToHost);
		/*for (int j = 0; j < NBPAR; ++j)
		{
			fprintf(fichier, 
			"%d %g %g %g\n",
			j, s->pos[j], s->pos[j+NBPAR], s->pos[j+2*NBPAR]);
		}
		if(i!= NBITER -1)
			fprintf(fichier, "\n\n");*/
	}

	/*pset_print(s);*/
	

	fclose(fichier);
	pset_free(s);
	hipFree(nb);
	hipFree(acc);
	hipFree(spd);
	hipFree(pos);
	hipFree(m);
	return 0;
}